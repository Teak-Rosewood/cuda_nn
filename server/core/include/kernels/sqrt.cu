#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "kernels.hpp"

template <typename T>
__global__ void sqrtKernel(T** a, float** b, int rows, int cols)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows && col < cols)
    {
        b[row][col] = sqrtf(float(a[row][col]));
    }
}

template <typename T>
void CUDAsqrt(T** device_data, float**& device_data_sqrt, std::pair<int, int> size) {
    // Allocate memory for the sqrt data
    hipMalloc(&device_data_sqrt, size.first * sizeof(float *));
    for (int i = 0; i < size.first; ++i) {
        hipMalloc(&device_data_sqrt[i], size.second * sizeof(float));
    }

    // Set up the grid and block dimensions
    dim3 blockSize(16, 16);
    dim3 gridSize((size.second + blockSize.x - 1) / blockSize.x, (size.first + blockSize.y - 1) / blockSize.y);

    // Launch the sqrtKernel
    sqrtKernel<T><<<gridSize, blockSize>>>(device_data, device_data_sqrt, size.first, size.second);
}