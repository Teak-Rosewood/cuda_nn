#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "kernels.hpp"

template <typename T>
__global__ void scalarMultiplyKernel(T** a, T multiplicand, T** c, int rows, int cols)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows && col < cols)
    {
        c[row][col] = a[row][col] * multiplicand;
    }
}

template <typename T>
void CUDAscalarMultiply(T** device_data, float multiplicand, T**& device_data_multiplied, std::pair<int, int> size) {
    device_data_multiplied = new T*[size.first];
    for (int i = 0; i < size.first; ++i) {
        hipMalloc(&device_data_multiplied[i], size.second * sizeof(T));
        hipMemset(device_data_multiplied[i], 0, size.second * sizeof(T));
    }

    dim3 blockSize(16, 16);
    dim3 gridSize((size.second + blockSize.x - 1) / blockSize.x, (size.first + blockSize.y - 1) / blockSize.y);

    scalarMultiplyKernel<T><<<gridSize, blockSize>>>(device_data, multiplicand, device_data_multiplied, size.first, size.second);
    hipDeviceSynchronize(); // Wait for CUDA to finish
}

template void CUDAscalarMultiply<float>(float** device_data, float multiplicand, float**& device_data_multiplied, std::pair<int, int> size);