#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "kernels.hpp"

template <typename T>
__global__ void reshapeKernel(T** a, T** b, int oldCols, int newRows, int newCols)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < newRows * newCols)
    {
        b[index / newCols][index % newCols] = a[index / oldCols][index % oldCols];
    }
}

template <typename T>
void CUDAreshape(T** device_data, T**& device_data_reshaped, std::pair<int, int> oldSize, std::pair<int, int> newSize) {
    // Allocate memory for the reshaped data
    hipMalloc(&device_data_reshaped, newSize.first * sizeof(T *));
    for (int i = 0; i < newSize.first; ++i) {
        hipMalloc(&device_data_reshaped[i], newSize.second * sizeof(T));
    }

    // Set up the grid and block dimensions
    int numThreads = 256;
    int numBlocks = (newSize.first * newSize.second + numThreads - 1) / numThreads;

    // Launch the reshapeKernel
    reshapeKernel<T><<<numBlocks, numThreads>>>(device_data, device_data_reshaped, oldSize.second, newSize.first, newSize.second);
}